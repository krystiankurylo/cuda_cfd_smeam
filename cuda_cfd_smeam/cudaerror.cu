#include "cudaerror.h"

//w przypadku jakiegokolwiek b��du w operacjach cuda wyrzuca w konsoli b��d i ko�czy dzia�anie programu
void RaiseError(hipError_t cudaStatus)
{
	if (cudaStatus != hipSuccess)
	{
		printf("error, %s", hipGetErrorString(cudaStatus));
		exit(-1);
	}
}