#include "hip/hip_runtime.h"
// written by Szymon Winczewski

#include "cuda_central_forces_file.h"

using namespace std;


CudaCentralForcesFile::CudaCentralForcesFile()
{
    mode_ = 0;

    memory_allocated_ = 0;
    number_of_atoms_ = 0;
    max_number_of_central_forces_ = 0;
    number_of_central_forces_ = NULL;
    central_forces_ = NULL;
}


CudaCentralForcesFile::~CudaCentralForcesFile()
{
    deallocateMemory();
}


void CudaCentralForcesFile::checkAvailability()
{
    if ( memory_allocated_ == 0 )
        raiseError(ERR_CENTRAL_FORCES_FILE, 1, "checkAvailability", "the requested operation is forbidden");
}


void CudaCentralForcesFile::allocateMemory()
{
    if ( memory_allocated_ == 0 )
    {
        allocateCentralForces(number_of_atoms_,
                              max_number_of_central_forces_,
                              number_of_central_forces_,
                              central_forces_);
        memory_allocated_ = 1;
    }
}


void CudaCentralForcesFile::deallocateMemory()
{
    if ( memory_allocated_ == 1 )
    {
        deallocateCentralForces(number_of_atoms_,
                                number_of_central_forces_,
                                central_forces_);
        memory_allocated_ = 0;
    }
}


void CudaCentralForcesFile::openOutFile(std::string file_name)
{
    if ( mode_ != 0 )
        raiseError(ERR_CENTRAL_FORCES_FILE, 2, "openOutFile", "the requested operation is forbidden");
    file_name_ = file_name;
    ofile_.open(file_name_.c_str(), std::ios::binary | std::ios::out);
    openOutputFileError(ERR_CENTRAL_FORCES_FILE, 3, "openOutFile", file_name_, ofile_);
    mode_ = 1;
}


void CudaCentralForcesFile::closeOutFile()
{
    if ( mode_ != 1 )
        raiseError(ERR_CENTRAL_FORCES_FILE, 4, "closeOutFile", "the requested operation is forbidden");
    ofile_.close();
    ofile_.clear();
    mode_ = 3;
}


void CudaCentralForcesFile::writeToOutFile(int number_of_atoms,
                                       int max_number_of_central_forces,
                                       int *number_of_central_forces,
                                       central_force **central_forces)
{
    if ( mode_ != 1 )
        raiseError(ERR_CENTRAL_FORCES_FILE, 5, "writeToOutFile", "the requested operation is forbidden");

    int i, j;
    central_force *this_central_force;

    ofile_.write((char*)&number_of_atoms, sizeof(int));
    ofile_.write((char*)&max_number_of_central_forces, sizeof(int));
    for (i = 0; i < number_of_atoms; i++)
    {
        ofile_.write((char*)&number_of_central_forces[i], sizeof(int));
        for (j = 0; j < number_of_central_forces[i]; j++)
        {
            this_central_force = &central_forces[i][j];
            ofile_.write((char*)&this_central_force->first_second, sizeof(bool));
            ofile_.write((char*)&this_central_force->atom_j, sizeof(int));
            ofile_.write((char*)&this_central_force->force[0], sizeof(double));
            ofile_.write((char*)&this_central_force->force[1], sizeof(double));
            ofile_.write((char*)&this_central_force->force[2], sizeof(double));
            ofile_.write((char*)&this_central_force->r_ij, sizeof(double));
            ofile_.write((char*)&this_central_force->r_ij_dir[0], sizeof(double));
            ofile_.write((char*)&this_central_force->r_ij_dir[1], sizeof(double));
            ofile_.write((char*)&this_central_force->r_ij_dir[2], sizeof(double));
        }
    }
}


void CudaCentralForcesFile::openInFile(std::string file_name)
{
    if ( mode_ != 0 )
        raiseError(ERR_CENTRAL_FORCES_FILE, 6, "openInFile", "the requested operation is forbidden");
    file_name_ = file_name;
    ifile_.open(file_name_.c_str(), std::ios::binary | std::ios::in);
    openInputFileError(ERR_CENTRAL_FORCES_FILE, 7, "openInFile", file_name_, ifile_);
    mode_ = 2;
}


void CudaCentralForcesFile::closeInFile()
{
    if ( mode_ != 2 )
        raiseError(ERR_CENTRAL_FORCES_FILE, 8, "closeInFile", "the requested operation is forbidden");
    ifile_.close();
    ifile_.clear();
    mode_ = 3;
}


void CudaCentralForcesFile::readFromInFile()
{
    if ( mode_ != 2 )
        raiseError(ERR_CENTRAL_FORCES_FILE, 9, "readFromInFile", "the requested operation is forbidden");

    int i, j;
    central_force *this_central_force;
    int tmp_number_of_atoms;
    int tmp_max_number_of_central_forces;

    ifile_.read((char*)&tmp_number_of_atoms, sizeof(int));
    if ( ifile_.eof() == 1 )
        raiseError(ERR_CENTRAL_FORCES_FILE, 10, "readFromInFile", "end of file reached");

    ifile_.read((char*)&tmp_max_number_of_central_forces, sizeof(int));
    if ( ifile_.eof() == 1 )
        raiseError(ERR_CENTRAL_FORCES_FILE, 10, "readFromInFile", "end of file reached");

    if ( memory_allocated_ == 0 )
    {
        number_of_atoms_ = tmp_number_of_atoms;
        max_number_of_central_forces_ = tmp_max_number_of_central_forces;
        allocateMemory();
    }
    else
    {
        if ( tmp_number_of_atoms != number_of_atoms_ )
            raiseError(ERR_CENTRAL_FORCES_FILE, 11, "readFromInFile", "number of atoms has changed");
        if ( tmp_max_number_of_central_forces != max_number_of_central_forces_ )
            raiseError(ERR_CENTRAL_FORCES_FILE, 12, "readFromInFile", "max number of central forces has changed");
        number_of_atoms_ = tmp_number_of_atoms;
        max_number_of_central_forces_ = tmp_max_number_of_central_forces;
    }

    for (i = 0; i < number_of_atoms_; i++)
    {
        ifile_.read((char*)&number_of_central_forces_[i], sizeof(int));
        if ( ifile_.eof() == 1 )
            raiseError(ERR_CENTRAL_FORCES_FILE, 10, "readFromInFile", "end of file reached");

        for (j = 0; j < number_of_central_forces_[i]; j++)
        {
            this_central_force = &central_forces_[i][j];
            ifile_.read((char*)&this_central_force->first_second, sizeof(bool));
            ifile_.read((char*)&this_central_force->atom_j, sizeof(int));
            ifile_.read((char*)&this_central_force->force[0], sizeof(double));
            ifile_.read((char*)&this_central_force->force[1], sizeof(double));
            ifile_.read((char*)&this_central_force->force[2], sizeof(double));
            ifile_.read((char*)&this_central_force->r_ij, sizeof(double));
            ifile_.read((char*)&this_central_force->r_ij_dir[0], sizeof(double));
            ifile_.read((char*)&this_central_force->r_ij_dir[1], sizeof(double));
            ifile_.read((char*)&this_central_force->r_ij_dir[2], sizeof(double));
        }
    }
}


int CudaCentralForcesFile::getNumberOfAtoms()
{
    checkAvailability();
    return number_of_atoms_;
}


int CudaCentralForcesFile::getMaxNumberOfCentralForces()
{
    checkAvailability();
    return max_number_of_central_forces_;
}


int *CudaCentralForcesFile::getNumberOfCentralForces()
{
    checkAvailability();
    return number_of_central_forces_;
}


central_force **CudaCentralForcesFile::getCentralForces()
{
    checkAvailability();
    return central_forces_;
}
