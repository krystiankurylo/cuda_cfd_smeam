#include "hip/hip_runtime.h"
#include <stdio.h>


#include "cudaspline.cuh"



#include <windows.h>
#include <time.h>
#include "cudaerror.h"

//kernel obliczania pochodnej
__global__ void kernelEvaluateDeriv(double* result,double* array_x,int size,double xmin_,double deriv0_,double derivN_,double h_,double hsq_,double xmax_shifted_,values_struct* values_vec_)
{
	int tidx = threadIdx.x + blockDim.x * blockIdx.x;

	if (tidx < size)
	{
		double x = array_x[tidx];

		x -= xmin_;
		if (x <= 0.0)
		{
			result[tidx] = deriv0_;
		}
		else if (x >= xmax_shifted_)
		{
			result[tidx] = derivN_;
		}
			
		else
		{
			int k = (int)(x / h_);
			double a = values_vec_[k].Xs_next - x;
			double b = h_ - a;
			double asq = a * a;
			double bsq = b * b;
			result[tidx] = values_vec_[k].Ydelta + ((3.0 * bsq - hsq_) * values_vec_[k].Y2_next - (3.0 * asq - hsq_) * values_vec_[k].Y2);
		}
	}
}


//obliczanie pochodnej
void CudaSpline::EvaluateDerivCUDA(double* result,double *array_x,int size)
{
	hipError_t cudaStatus;

	dim3 block(1024, 1024);
	dim3 grid(32, 32);

	double* dev_result;
	double* dev_array_x;
	values_struct* dev_values_vec;

	//alokowanie pami�ci dla wyniku
	cudaStatus = hipMalloc(&dev_result, size*sizeof(double));

	RaiseError(cudaStatus);

	//alokowanie pami�ci dla dev_array_x
	cudaStatus = hipMalloc(&dev_array_x, size*sizeof(double));

	RaiseError(cudaStatus);

	//kopiowaine pami�ci dla dev_array_x
	cudaStatus = hipMemcpy(dev_array_x, array_x, size*sizeof(double), hipMemcpyHostToDevice);

	RaiseError(cudaStatus);

	//alokowanie pami�ci dla dev_values_vec

	cudaStatus = hipMalloc(&dev_values_vec,N_*sizeof(values_struct));

	RaiseError(cudaStatus);

	cudaStatus = hipMemcpy(dev_values_vec, values_vec_, N_*sizeof(values_struct), hipMemcpyHostToDevice);

	RaiseError(cudaStatus);

	kernelEvaluateDeriv<<<block,grid>>>(dev_result, dev_array_x, size, xmin_, deriv0_, derivN_, h_, hsq_, xmax_shifted_, dev_values_vec);

	//kopiowanie pami�ci dla result
	cudaStatus = hipMemcpy(result, dev_result, size*sizeof(double), hipMemcpyDeviceToHost);

	RaiseError(cudaStatus);

	hipFree(dev_values_vec);
	hipFree(dev_array_x);
	hipFree(dev_result);
}

CudaSpline::CudaSpline()
{
	X_ = NULL;
	Xs_ = NULL;
	Y_ = NULL;
	Y2_ = NULL;
	Ydelta_ = NULL;
	N_ = 0;
	value0_ = 0.0;
	deriv0_ = 0.0;
	valueN_ = 0.0;
	derivN_ = 0.0;
	xmin_ = 0.0;
	xmax_ = 0.0;
	cutoff_ = 0.0;
	h_ = 0.0;
	hsq_ = 0.0;
	xmax_shifted_ = 0.0;
#ifdef SPLINE_FAST
	values_vec_ = NULL;
#endif
}
CudaSpline::~CudaSpline()
{
	if (X_ != NULL)
		delete[] X_;
	if (Xs_ != NULL)
		delete[] Xs_;
	if (Y_ != NULL)
		delete[] Y_;
	if (Y2_ != NULL)
		delete[] Y2_;
	if (Ydelta_ != NULL)
		delete[] Ydelta_;
#ifdef SPLINE_FAST
	if (values_vec_ != NULL)
		delete[] values_vec_;
#endif
}

double CudaSpline::getCutoff()
{
	return cutoff_;
}

//stary evaluate - nieu�ywany
__global__ void evaluateKernel(pair* results, int N, double x, double xmin_, double h_, double xmax_shifted_, double dx, double value0_, double deriv0_, double valueN_, double derivN_, double hsq_, values_struct* values_vec_)
{
	int tid = blockIdx.x;
	if (tid < N)
	{
		x += tid*dx;
		results[tid].key = x;
		x -= xmin_;
		if (x <= 0.0)
		{
			results[tid].value = value0_ + deriv0_ * x;
		}
		else if (x >= xmax_shifted_)
		{
			results[tid].value = valueN_ + derivN_ * (x - xmax_shifted_);
		}
		else
		{
			int k = (int)(x / h_);
			double a = values_vec_[k].Xs_next - x;
			double b = h_ - a;
			results[tid].value = values_vec_[k].Y_next - a * values_vec_[k].Ydelta + ((a * a - hsq_) * a * values_vec_[k].Y2 + (b * b - hsq_) * b * values_vec_[k].Y2_next);
		}
	}
    
}

//kernel evaluate
__global__ void evaluateKernel(vec3d* n_bonds_1D, meam_bond* bonds_list_central_1D,int* n_num_, int number_of_atoms_, int max_number_of_n_neighbours_,double xmin_,double deriv0_,double derivN_,double value0_,double valueN_,double xmax_shifted_,values_struct *values_vec_,double h_,double hsq_)
{

	int tidx = threadIdx.x + blockDim.x*blockIdx.x;
	int tidy = threadIdx.y + blockDim.y*blockIdx.y;



	if (tidx < number_of_atoms_)
	{	
			int number_of_bonds = n_num_[tidx];
			if (tidy < number_of_bonds)
			{
				vec3d *this_bond = &n_bonds_1D[tidx*max_number_of_n_neighbours_ + tidy];
				double r_ij = this_bond->r;
				r_ij -= xmin_;
				if (r_ij <= 0.0)
				{
					bonds_list_central_1D[tidx*max_number_of_n_neighbours_ + tidy].fprime_r_ij = deriv0_;
					bonds_list_central_1D[tidx*max_number_of_n_neighbours_ + tidy].f_r_ij = value0_ + deriv0_ * r_ij;
				}
				else if (r_ij >= xmax_shifted_)
				{
					bonds_list_central_1D[tidx*max_number_of_n_neighbours_ + tidy].fprime_r_ij = derivN_;
					bonds_list_central_1D[tidx*max_number_of_n_neighbours_ + tidy].f_r_ij = valueN_ + derivN_ * (r_ij - xmax_shifted_);
				}
				else
				{
					// Xs_next, Ydelta, Y2_next, Y2, Y_next
					int k = (int)(r_ij / h_);
					double a = values_vec_[k].Xs_next - r_ij;
					double b = h_ - a;
					double asq = a * a;

					double bsq = b * b;

					//	printf("%lf:%lf\t",asq,bsq);
					bonds_list_central_1D[tidx*max_number_of_n_neighbours_ + tidy].fprime_r_ij = values_vec_[k].Ydelta + ((3.0 * bsq - hsq_) * values_vec_[k].Y2_next - (3.0 * asq - hsq_) * values_vec_[k].Y2);
					bonds_list_central_1D[tidx*max_number_of_n_neighbours_ + tidy].f_r_ij = values_vec_[k].Y_next - a * values_vec_[k].Ydelta + ((asq - hsq_) * a * values_vec_[k].Y2 + (bsq - hsq_) * b * values_vec_[k].Y2_next);
				}
			}
	}

}

//evaluate
void CudaSpline::evaluateCUDA(vec3d* n_bonds_1D,meam_bond* bonds_list_central_1D,int* n_num_,int number_of_atoms_,int max_number_of_n_neighbours_)
{
	vec3d *dev_n_bonds_1D;
	meam_bond *dev_bonds_list_central_1D;
	int * dev_n_num_;
	values_struct* dev_values_vec_;

	hipError_t cudaStatus;

	//alokowanie pami�ci dla values_vec
	cudaStatus = hipMalloc(&dev_values_vec_, N_*sizeof(values_struct));
	
	RaiseError(cudaStatus);

	//kopiowanie pami�ci dla values_vec
	cudaStatus = hipMemcpy(dev_values_vec_, values_vec_, N_*sizeof(values_struct), hipMemcpyHostToDevice);

	RaiseError(cudaStatus);

	//alokowanie pami�ci dla n_bonds_1D
	cudaStatus = hipMalloc(&dev_n_bonds_1D, number_of_atoms_*max_number_of_n_neighbours_*sizeof(vec3d));

	RaiseError(cudaStatus);

	//kopiowanie pami�ci dla n_bonds_1D
	cudaStatus = hipMemcpy(dev_n_bonds_1D, n_bonds_1D, number_of_atoms_*max_number_of_n_neighbours_*sizeof(vec3d), hipMemcpyHostToDevice);

	RaiseError(cudaStatus);

	//alokowanie pami�ci dla bonds_list_central_1D
	cudaStatus = hipMalloc(&dev_bonds_list_central_1D, max_number_of_n_neighbours_*number_of_atoms_*sizeof(meam_bond));

	RaiseError(cudaStatus);

	cudaStatus = hipMemcpy(dev_bonds_list_central_1D, bonds_list_central_1D, max_number_of_n_neighbours_*number_of_atoms_*sizeof(meam_bond),hipMemcpyHostToDevice);

	RaiseError(cudaStatus);

	//alokowanie pami�ci dla n_num
	cudaStatus = hipMalloc(&dev_n_num_, number_of_atoms_*sizeof(int));
	
	RaiseError(cudaStatus);

	//kopiowanie pami�ci dla n_num
	cudaStatus = hipMemcpy(dev_n_num_, n_num_, number_of_atoms_*sizeof(int), hipMemcpyHostToDevice);

	RaiseError(cudaStatus);

	dim3 block(1024, 1024);
	dim3 grid(32,32);

	evaluateKernel << <block, grid >> >(dev_n_bonds_1D, dev_bonds_list_central_1D, dev_n_num_, number_of_atoms_, max_number_of_n_neighbours_, xmin_ , deriv0_, derivN_, value0_, valueN_, xmax_shifted_, dev_values_vec_, h_,hsq_);
	
	//kopiowanie pami�ci dla bonds_list_central_1D
	cudaStatus = hipMemcpy(bonds_list_central_1D, dev_bonds_list_central_1D, max_number_of_n_neighbours_*number_of_atoms_*sizeof(meam_bond), hipMemcpyDeviceToHost);

	RaiseError(cudaStatus);

	hipFree(dev_bonds_list_central_1D);
	hipFree(dev_n_bonds_1D);
	hipFree(dev_values_vec_);
	hipFree(dev_n_num_);
}

//stary evaluate
double CudaSpline::evaluateCUDA(pair* results,double x,double dx,int N)
{
	values_struct* dev_values_vec = 0;
	pair *dev_results = 0;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsedTime = 0;

	hipEventRecord(start, 0);

	hipError_t cudaStatus;
	


	cudaStatus = hipMalloc((void**)&dev_results, N_*sizeof(pair));
	cudaStatus = hipMalloc((void**)&dev_values_vec, N_*sizeof(values_struct));

	cudaStatus = hipMemcpy(dev_values_vec, values_vec_, N_*sizeof(values_struct), hipMemcpyHostToDevice);

	evaluateKernel <<<N_,1>>>(dev_results, N_, x,xmin_, h_, xmax_shifted_, dx, value0_, deriv0_, valueN_, derivN_, hsq_, dev_values_vec);


	cudaStatus = hipMemcpy(results, dev_results, N_*sizeof(pair), hipMemcpyDeviceToHost);

	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "evaluateCUDA failed!");
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipFree(dev_results);
	hipFree(dev_values_vec);

	return (double)elapsedTime;
}

void CudaSpline::initialize(int N, double deriv0, double derivN,
	double xmin, double xmax, double *Y)
{
	int i;

	N_ = N;
	if ((N_ < 3) || (N_ != N_))
		raiseError(ERR_SPLINE, 15, "initialize", "incorrect N");

	deriv0_ = deriv0;
	derivN_ = derivN;

	xmin_ = xmin;
	xmax_ = xmax;
	if (xmin_ >= xmax_)
		raiseError(ERR_SPLINE, 16, "initialize", "incorrect xmin and xmax");

	cutoff_ = xmax_;
	h_ = (xmax_ - xmin_) / (double(N_) - 1.0);
	hsq_ = h_ * h_;

	if (Y == NULL)
		raiseError(ERR_SPLINE, 17, "initialize", "NULL pointer passed as an argument");

	X_ = new double[N_];
	Y_ = new double[N_];
	for (i = 0; i < N_; i++)
	{
		X_[i] = xmin_ + i * h_;
		Y_[i] = Y[i];

		if (Y_[i] != Y_[i])
			raiseError(ERR_SPLINE, 18, "initialize", "incorrect Y[i]");
	}

	init();
}

void CudaSpline::init()
{
	int i, k;
	double sig, p;
	double qn, un;

	xmax_shifted_ = xmax_ - xmin_;

	Xs_ = new double[N_];
	Ydelta_ = new double[N_];
	Y2_ = new double[N_];


	for (i = 0; i < N_; i++)
		Xs_[i] = i * h_;

	double *u = new double[N_];
	Y2_[0] = -0.5;
	u[0] = (3.0 / (X_[1] - X_[0])) * ((Y_[1] - Y_[0]) / (X_[1] - X_[0]) - deriv0_);
	for (i = 1; i < (N_ - 1); i++)
	{
		sig = (X_[i] - X_[i - 1]) / (X_[i + 1] - X_[i - 1]);
		p = sig * Y2_[i - 1] + 2.0;
		Y2_[i] = (sig - 1.0) / p;
		u[i] = (Y_[i + 1] - Y_[i]) / (X_[i + 1] - X_[i]) - (Y_[i] - Y_[i - 1]) / (X_[i] - X_[i - 1]);
		u[i] = (6.0 * u[i] / (X_[i + 1] - X_[i - 1]) - sig * u[i - 1]) / p;
	}

	qn = 0.5;
	un = (3.0 / (X_[N_ - 1] - X_[N_ - 2])) * (derivN_ - (Y_[N_ - 1] - Y_[N_ - 2]) / (X_[N_ - 1] - X_[N_ - 2]));
	Y2_[N_ - 1] = (un - qn *u[N_ - 2]) / (qn * Y2_[N_ - 2] + 1.0);
	for (k = N_ - 2; k >= 0; k--)
		Y2_[k] = Y2_[k] * Y2_[k + 1] + u[k];

	delete[] u;

	for (i = 0; i < N_; i++)
	{
		if (i < (N_ - 1))
			Ydelta_[i] = (Y_[i + 1] - Y_[i]) / h_;
		Y2_[i] /= (h_ * 6.0);
	}

	value0_ = Y_[0];
	valueN_ = Y_[N_ - 1];

#ifdef SPLINE_FAST
	values_vec_ = new values_struct[N_];
	for (i = 0; i < N_ - 1; i++)
	{
		values_vec_[i].Xs_next = Xs_[i + 1];
		values_vec_[i].Y_next = Y_[i + 1];
		values_vec_[i].Ydelta = Ydelta_[i];
		values_vec_[i].Y2 = Y2_[i];
		values_vec_[i].Y2_next = Y2_[i + 1];
	}
#endif

}

void CudaSpline::ShowParams()
{
	std::cout << "N      = " << N_ << std::endl;
	std::cout << "value0 = " << std::setprecision(14) << value0_ << std::endl;
	std::cout << "deriv0 = " << std::setprecision(14) << deriv0_ << std::endl;
	std::cout << "valueN = " << std::setprecision(14) << valueN_ << std::endl;
	std::cout << "derivN = " << std::setprecision(14) << derivN_ << std::endl;
	std::cout << "xmin   = " << std::setprecision(14) << xmin_ << std::endl;
	std::cout << "xmax   = " << std::setprecision(14) << xmax_ << std::endl;
	std::cout << "values: " << std::endl;
	for (int i = 0; i < N_; i++)
		std::cout << std::setprecision(14) << Y_[i] << std::endl;
}

void CudaSpline::readFromASCIIFile(std::ifstream &input)
{
	int i;
	std::string keyword;

	input >> keyword;
	input >> N_;
	if (keyword != "N")
		raiseError(ERR_SPLINE, 1, "readFromASCIIFile", "\"N\" was expected");
	if ((N_ < 3) || (N_ != N_))
		raiseError(ERR_SPLINE, 2, "readFromASCIIFile", "incorrect N");

	input >> keyword;
	input >> deriv0_;
	if (keyword != "deriv0")
		raiseError(ERR_SPLINE, 3, "readFromASCIIFile", "\"deriv0\" was expected");
	if (deriv0_ != deriv0_)
		raiseError(ERR_SPLINE, 4, "readFromASCIIFile", "incorrect deriv0");

	input >> keyword;
	input >> derivN_;
	if (keyword != "derivN")
		raiseError(ERR_SPLINE, 5, "readFromASCIIFile", "\"derivN\" was expected");
	if (derivN_ != derivN_)
		raiseError(ERR_SPLINE, 6, "readFromASCIIFile", "incorrect derivN");

	input >> keyword;
	input >> xmin_;
	if (keyword != "xmin")
		raiseError(ERR_SPLINE, 7, "readFromASCIIFile", "\"xmin\" was expected");
	if (xmin_ != xmin_)
		raiseError(ERR_SPLINE, 8, "readFromASCIIFile", "incorrect xmin");

	input >> keyword;
	input >> xmax_;
	if (keyword != "xmax")
		raiseError(ERR_SPLINE, 9, "readFromASCIIFile", "\"xmax\" was expected");
	if (xmax_ != xmax_)
		raiseError(ERR_SPLINE, 10, "readFromASCIIFile", "incorrect xmax");

	if (xmin_ >= xmax_)
		raiseError(ERR_SPLINE, 11, "readFromASCIIFile", "incorrect xmin and xmax");

	cutoff_ = xmax_;
	h_ = (xmax_ - xmin_) / (double(N_) - 1.0);
	hsq_ = h_ * h_;

	input >> keyword;
	if (keyword != "values")
		raiseError(ERR_SPLINE, 12, "readFromASCIIFile", "\"values\" was expected");

	X_ = new double[N_];
	Y_ = new double[N_];
	for (i = 0; i < N_; i++)
	{
		X_[i] = xmin_ + i * h_;
		input >> Y_[i];

		if (Y_[i] != Y_[i])
			raiseError(ERR_SPLINE, 13, "readFromASCIIFile", "incorrect Y[i]");

		if (input.eof() == 1)
			raiseError(ERR_SPLINE, 14, "readFromASCIIFile", "unexpected end of file");
	}

	init();
}




