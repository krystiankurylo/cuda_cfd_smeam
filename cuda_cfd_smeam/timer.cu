#include "hip/hip_runtime.h"
// written by Szymon Winczewski

#include "timer.h"

using namespace std;


Timer::Timer(std::string timer_name, int max_number_of_routines)
{
    if ( timer_name == "" )
        raiseError(ERR_TIMER, 1, "Timer", "wrong timer name specified");
    timer_name_ = timer_name;

    if ( max_number_of_routines < 0 )
        raiseError(ERR_TIMER, 2, "Timer", "incorrect maximum number of routines parameter");
    max_number_of_routines_ = max_number_of_routines;

    routine_defined_     = new bool [max_number_of_routines_];
    for (int i = 0; i < max_number_of_routines_; i++)
        routine_defined_[i] = 0;
    routine_name_        = new std::string[max_number_of_routines_];
    routine_executions_  = new long int [max_number_of_routines_];
    routine_total_time_  = new double [max_number_of_routines_];
    routine_av_time_     = new double [max_number_of_routines_];
    routine_total_clock_ = new clock_t [max_number_of_routines_];
    routine_started_     = new clock_t [max_number_of_routines_];
    routine_stopped_     = new clock_t [max_number_of_routines_];
}


Timer::~Timer()
{
    delete [] routine_defined_;
    delete [] routine_name_;
    delete [] routine_executions_;
    delete [] routine_total_time_;
    delete [] routine_av_time_;
    delete [] routine_total_clock_;
    delete [] routine_started_;
    delete [] routine_stopped_;
}


void Timer::resetTimings()
{
    for (int i = 0; i < max_number_of_routines_; i++)
    {
        routine_executions_[i] = 0;
        routine_total_time_[i] = 0.0;
        routine_av_time_[i] = 0.0;
        routine_total_clock_[i] = 0;
    }
}


void Timer::addRoutine(std::string routine_name, int routine_id)
{
    if ( ( routine_id < 0 ) || ( routine_id > max_number_of_routines_ ) )
        raiseError(ERR_TIMER, 3, "addRoutine", "wrong routine id");
    if ( routine_defined_[routine_id] == 1 )
        raiseError(ERR_TIMER, 4, "addRoutine", "routine already defined");

    routine_defined_[routine_id] = 1;
    routine_name_[routine_id] = routine_name;
    routine_executions_[routine_id] = 0;
    routine_total_time_[routine_id] = 0.0;
    routine_av_time_[routine_id] = 0.0;
    routine_total_clock_[routine_id] = 0;
}


void Timer::routineStarted(int routine_id)
{
    routine_started_[routine_id] = clock();
}


void Timer::routineStopped(int routine_id)
{
    routine_stopped_[routine_id] = clock();
    routine_total_clock_[routine_id] += ( routine_stopped_[routine_id] - routine_started_[routine_id] );
    routine_executions_[routine_id]++;
}


void Timer::printTimings()
{
    std::cout << std::endl;
    std::cout << "Timer: " << timer_name_ << std::endl;
    for (int i = 0; i < max_number_of_routines_; i++)
    {
        if ( routine_defined_[i] == 1 )
        {
            routine_total_time_[i] = double(routine_total_clock_[i]) / double(CLOCKS_PER_SEC);
            if ( routine_executions_[i] != 0 )
                routine_av_time_[i] = double(routine_total_time_[i]) / double(routine_executions_[i]);
            else
                routine_av_time_[i] = 0.0;

            std::cout << "   " << setw(50) << routine_name_[i] << " / "
                          << setw(10) << routine_executions_[i] << " / "
                          << setw(15)  << setprecision(9) << routine_total_time_[i] << " / "
                          << setw(15) << setprecision(9) << routine_av_time_[i] << " /" << std::endl;
        }
    }
    std::cout << std::endl;
    std::cout << std::endl;
}
