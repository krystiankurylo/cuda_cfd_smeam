#include "hip/hip_runtime.h"
// written by Szymon Winczewski

#include "cuda_cfd_smeam.h"

#include <hip/hip_runtime.h>
#include <>

#include "cudaerror.h"

using namespace std;

//__global__ void kernelComputeElectronDensity(int* number_of_bonds_central_,meam_bond* bonds_list_central_,int number_of_atoms_,int max_number_of_n_neighbours_)
//{
//	double n_i_total = 0.0;
//	double n_i_three_body;
//	double cos_theta_jik;
//
//	meam_bond bond_ij;
//	meam_bond bond_ik;
//
//	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
//	int tidy = threadIdx.y + blockIdx.y * blockDim.y;
//
//	if (tidx < number_of_atoms_)
//	{
//		int number_of_bonds = number_of_bonds_central_[tidx]; 
//
//		if (tidy < number_of_bonds)
//		{
//			memcpy(&bond_ij, &bonds_list_central_[tidx*max_number_of_n_neighbours_ + tidy], sizeof(meam_bond));
//
//			n_i_three_body = 0;
//
//			
//			cos_theta_jik = bond_ij.r_ij_dir[0] * bond_ik.r_ij_dir[0] + bond_ij.r_ij_dir[1] * bond_ik.r_ij_dir[1] + bond_ij.r_ij_dir[2] * bond_ik.r_ij_dir[2];
//		}
//		
//	}
//
//}

//tworzenie listy wi�za�
__global__ void kernelCreateBondsList(meam_bond* bonds_list_central_,int *n_list,int *n_num_,vec3d *n_bonds_, int number_of_atoms_, int max_number_of_n_neighbours_,int* number_of_bonds_central)
{
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y;

	if (tidx < number_of_atoms_)
	{

		if (tidy < n_num_[tidx])
		{
			number_of_bonds_central[tidx] = n_num_[tidx];

			bonds_list_central_[tidx*max_number_of_n_neighbours_ + tidy].atom_j = n_list[tidx*max_number_of_n_neighbours_ + tidy];
			bonds_list_central_[tidx*max_number_of_n_neighbours_ + tidy].r_ij = n_bonds_[tidx*max_number_of_n_neighbours_ + tidy].r;

			double inv_of_r_ij = 1.0 / n_bonds_[tidx*max_number_of_n_neighbours_ + tidy].r;

			bonds_list_central_[tidx*max_number_of_n_neighbours_ + tidy].inv_of_r_ij = inv_of_r_ij;
			bonds_list_central_[tidx*max_number_of_n_neighbours_ + tidy].r_ij_dir[0] = inv_of_r_ij * n_bonds_[tidx*max_number_of_n_neighbours_ + tidy].vec[0];
			bonds_list_central_[tidx*max_number_of_n_neighbours_ + tidy].r_ij_dir[0] = inv_of_r_ij * n_bonds_[tidx*max_number_of_n_neighbours_ + tidy].vec[1];
			bonds_list_central_[tidx*max_number_of_n_neighbours_ + tidy].r_ij_dir[0] = inv_of_r_ij * n_bonds_[tidx*max_number_of_n_neighbours_ + tidy].vec[2];
		}
		
	}
	

}

//t�umaczenie tablic dwuwymiariowych na jednowymiarowe

void CUDA_CFD_sMEAM::Convert2DTo1DArray(int** source, int* destination)
{
	for (int i = 0; i < number_of_atoms_; i++)
	{
		for (int j = 0; j < max_number_of_n_neighbours_; j++)
		{
			destination[i*max_number_of_n_neighbours_ + j] = source[i][j];
		}
	}
}

void CUDA_CFD_sMEAM::Convert2DTo1DArray(vec3d** source, vec3d* destination)
{
	//printf("wtf\n");

	for (int i = 0; i < number_of_atoms_; i++)
	{
		for (int j = 0; j < max_number_of_n_neighbours_; j++)
		{
			//printf("%lf,%lf,%lf,%lf ", source[i][j].r, source[i][j].vec[0], source[i][j].vec[1], source[i][i].vec[2]);
			//system("pause");
			destination[i*max_number_of_n_neighbours_ + j].r = source[i][j].r;
			destination[i*max_number_of_n_neighbours_ + j].vec[0] = source[i][j].vec[0];
			destination[i*max_number_of_n_neighbours_ + j].vec[1] = source[i][j].vec[1];
			destination[i*max_number_of_n_neighbours_ + j].vec[2] = source[i][j].vec[2];
		}
	}
}
void CUDA_CFD_sMEAM::Convert2DTo1DArray(meam_bond** source,meam_bond* destination)
{
	for (int i = 0; i < number_of_atoms_; i++)
	{
		for (int j = 0; j < n_num_[i]; j++)
		{
			destination[i*max_number_of_n_neighbours_ + j].atom_j = source[i][j].atom_j;
			destination[i*max_number_of_n_neighbours_ + j].fprime_r_ij = source[i][j].fprime_r_ij;
			destination[i*max_number_of_n_neighbours_ + j].f_r_ij = source[i][j].f_r_ij;
			destination[i*max_number_of_n_neighbours_ + j].inv_of_r_ij = source[i][j].inv_of_r_ij;
			destination[i*max_number_of_n_neighbours_ + j].r_ij = source[i][j].r_ij;
			destination[i*max_number_of_n_neighbours_ + j].r_ij_dir[0] = source[i][j].r_ij_dir[0];
			destination[i*max_number_of_n_neighbours_ + j].r_ij_dir[1] = source[i][j].r_ij_dir[1];
			destination[i*max_number_of_n_neighbours_ + j].r_ij_dir[2] = source[i][j].r_ij_dir[2];
		}
	}
}

//t�umaczenie tablic jednowymiarowych na dwuwymiarowe

void CUDA_CFD_sMEAM::Convert1DTo2DArray(meam_bond* source,meam_bond** destination)
{
	for (int i = 0; i < number_of_atoms_; i++)
	{
		for (int j = 0; j < n_num_[i]; j++)
		{
			destination[i][j].atom_j = source[i*max_number_of_n_neighbours_ + j].atom_j;
			destination[i][j].fprime_r_ij = source[i*max_number_of_n_neighbours_ + j].fprime_r_ij;
			destination[i][j].f_r_ij = source[i*max_number_of_n_neighbours_ + j].f_r_ij;
			destination[i][j].inv_of_r_ij = source[i*max_number_of_n_neighbours_ + j].inv_of_r_ij;
			destination[i][j].r_ij = source[i*max_number_of_n_neighbours_ + j].r_ij;
			destination[i][j].r_ij_dir[0] = source[i*max_number_of_n_neighbours_ + j].r_ij_dir[0];
			destination[i][j].r_ij_dir[1] = source[i*max_number_of_n_neighbours_ + j].r_ij_dir[1];
			destination[i][j].r_ij_dir[2] = source[i*max_number_of_n_neighbours_ + j].r_ij_dir[2];
		}
	}
}

void CUDA_CFD_sMEAM::DoStartWithCuda()
{
	hipError_t cudaStatus;

	meam_bond *bonds_list_central_1D = (meam_bond*)malloc(max_number_of_n_neighbours_*number_of_atoms_*sizeof(meam_bond));

	int* dev_number_of_bonds_central;

	int *n_list_1D = (int*)malloc(number_of_atoms_*max_number_of_n_neighbours_*sizeof(int));
	vec3d *n_bonds_1D = (vec3d*)malloc(number_of_atoms_*max_number_of_n_neighbours_*sizeof(vec3d));

	Convert2DTo1DArray(n_bonds_, n_bonds_1D);
	Convert2DTo1DArray(n_list_, n_list_1D);
	Convert2DTo1DArray(bonds_list_central_, bonds_list_central_1D);

	vec3d *dev_n_bonds_1D;
	meam_bond *dev_bonds_list_central_1D;
	int *dev_n_num_;
	int *dev_n_list_1D;

	

	cudaStatus = hipMalloc(&dev_number_of_bonds_central, number_of_atoms_*sizeof(int));

	RaiseError(cudaStatus);

	cudaStatus = hipMalloc(&dev_n_num_, number_of_atoms_*sizeof(int));

	RaiseError(cudaStatus);

	cudaStatus = hipMemcpy(dev_n_num_, n_num_, number_of_atoms_*sizeof(int), hipMemcpyHostToDevice);

	RaiseError(cudaStatus);

	cudaStatus = hipMalloc(&dev_n_bonds_1D, number_of_atoms_*max_number_of_n_neighbours_*sizeof(vec3d));

	RaiseError(cudaStatus);

	cudaStatus = hipMemcpy(dev_n_bonds_1D, n_bonds_1D, number_of_atoms_*max_number_of_n_neighbours_*sizeof(vec3d), hipMemcpyHostToDevice);

	RaiseError(cudaStatus);

	cudaStatus = hipMalloc(&dev_bonds_list_central_1D, max_number_of_n_neighbours_*number_of_atoms_*sizeof(meam_bond));

	RaiseError(cudaStatus);

	cudaStatus = hipMalloc(&dev_n_list_1D, number_of_atoms_*max_number_of_n_neighbours_*sizeof(int));

	RaiseError(cudaStatus);

	cudaStatus = hipMemcpy(dev_n_list_1D, n_list_1D, max_number_of_n_neighbours_*number_of_atoms_*sizeof(int), hipMemcpyHostToDevice);

	RaiseError(cudaStatus);

	dim3 block(1024, 1024);
	dim3 grid(32, 32);

	kernelCreateBondsList << <block, grid >> >(dev_bonds_list_central_1D, dev_n_list_1D, dev_n_num_, dev_n_bonds_1D, number_of_atoms_, max_number_of_n_neighbours_, dev_number_of_bonds_central);
	
	cudaStatus = hipMemcpy(bonds_list_central_1D, dev_bonds_list_central_1D, max_number_of_n_neighbours_*number_of_atoms_*sizeof(meam_bond), hipMemcpyDeviceToHost);
	RaiseError(cudaStatus);
	
	//liczenie f i fprime
	f_spline_->evaluateCUDA(n_bonds_1D, bonds_list_central_1D, n_num_, number_of_atoms_, max_number_of_n_neighbours_);
	
	Convert1DTo2DArray(bonds_list_central_1D, bonds_list_central_);
	
	cudaStatus = hipMemcpy(number_of_bonds_central_, dev_number_of_bonds_central, number_of_atoms_*sizeof(int), hipMemcpyDeviceToHost);

	RaiseError(cudaStatus);
	
	double* n_i_total = (double*)malloc(number_of_atoms_*sizeof(double));

	for (int atom_i = 0; atom_i < number_of_atoms_; atom_i++)
	{
		n_i_total[atom_i] = computeElectronDensity(number_of_bonds_central_[atom_i], bonds_list_central_[atom_i]);
	}

	U_spline_->EvaluateDerivCUDA(Uprime_, n_i_total, number_of_atoms_);

	hipFree(dev_n_bonds_1D);
	hipFree(dev_bonds_list_central_1D);
	hipFree(dev_number_of_bonds_central);
	hipFree(dev_n_list_1D);
	hipFree(dev_n_num_);
	
}
int CUDA_CFD_sMEAM::createBondsList(int atom_i, meam_bond *bonds_list)
{
// tworzymy liste wiazan atomu i-tego,
// dla kazdego z sasiadow na liscie wiazan wyznaczamy i zapamietujemy:
// id sasiada, r_ij, r_ij_dir, f_r_ij, fprime_r_ij
    int jj;
    vec3d *this_bond;
    double r_ij, inv_of_r_ij;
	
    int number_of_bonds = n_num_[atom_i];
    for (jj = 0; jj  < number_of_bonds; jj++)
    {
        this_bond = &n_bonds_[atom_i][jj];
        bonds_list[jj].atom_j = n_list_[atom_i][jj];
        r_ij = this_bond->r;
        bonds_list[jj].r_ij = r_ij;
        inv_of_r_ij = 1.0 / r_ij;
        bonds_list[jj].inv_of_r_ij = inv_of_r_ij;
        bonds_list[jj].r_ij_dir[0] = inv_of_r_ij * this_bond->vec[0];
        bonds_list[jj].r_ij_dir[1] = inv_of_r_ij * this_bond->vec[1];
        bonds_list[jj].r_ij_dir[2] = inv_of_r_ij * this_bond->vec[2];
        bonds_list[jj].f_r_ij = f_spline_->evaluate(r_ij, bonds_list[jj].fprime_r_ij);
    }

    return number_of_bonds;
}


double CUDA_CFD_sMEAM::computeElectronDensity(int number_of_bonds, meam_bond *bonds_list)
{
// wyznaczamy gestosc n_i dana jako:
//    n_i = sum_j rho(r_ij) + 1/2 sum_jk f(r_ij) f(r_ik) g(cos_theta_jik)
    int j, k;
    double n_i_total = 0.0;
    double n_i_three_body;
    double cos_theta_jik;
    meam_bond bond_ij;
    meam_bond bond_ik;

    for (j = 0; j < number_of_bonds; j++)
    {
        memcpy(&bond_ij, &bonds_list[j], sizeof(meam_bond));

        n_i_three_body = 0.0;
        for (k = j + 1; k < number_of_bonds; k++)
        {
            memcpy(&bond_ik, &bonds_list[k], sizeof(meam_bond));

            computeCosThetaJIK(bond_ij.r_ij_dir, bond_ik.r_ij_dir, cos_theta_jik);

            n_i_three_body += bond_ik.f_r_ij * g_spline_->evaluate(cos_theta_jik);
        }

        n_i_total += bond_ij.f_r_ij * n_i_three_body;
        n_i_total += rho_spline_->evaluate(bond_ij.r_ij);
    }

    return n_i_total;
}


void CUDA_CFD_sMEAM::compute_central_forces()
{
    #ifdef CFD_SMEAM_TIMINGS
    timer_->routineStarted(3);
    #endif

    #ifdef CFD_SMEAM_DEBUG
    std::cout << "*** CFD_sMEAM::compute_central_forces() called! ***" << std::endl;
    #endif

    int i, j;
    int atom_i, atom_j, atom_k;
    int number_of_bonds_i, number_of_bonds_j;
    meam_bond *bonds_list_i, *bonds_list_j;
    double n_i_total;

    double Theta1_jik, Theta2_jik, Theta3_jik;
    double Theta1_ijk, Theta2_ijk, Theta3_ijk;

    double Theta4_kji;
    double r_ij_vec[3];

    bool nonzero_force;

    vec3d *bond_r_ij;
    double r_ij, inv_of_r_ij, r_ij_dir[3];
    double f_r_ij, fprime_r_ij;
    double inv_of_r_ik, r_ik_dir[3];
    double f_r_ik;

    double cos_theta_jik, g_cos_theta_jik, gprime_cos_theta_jik;
    double inv_of_r_jk, r_jk_dir[3];
    double f_r_jk;
    double cos_theta_kji, g_cos_theta_kji, gprime_cos_theta_kji;
    double prefactor;
    double phi_prime_r_ij, rho_prime_r_ij;

    central_force *this_central_force;
    int tmp_number_of_central_forces;
    double tmp_central_force[3];

// sprytna lista sasiadow - zerowanie listy
    int minus_one = -1;
    for (atom_i = 0; atom_i < number_of_atoms_; atom_i++)
        memcpy(&smart_neighbours_list_[atom_i][0], &minus_one, sizeof(int));

	DoStartWithCuda();

	// dla kazdego z atomow tworzymy liste wiazan oraz wyznaczamy n_i, U(n_i), U'(n_i)
	for (atom_i = 0; atom_i < number_of_atoms_; atom_i++)
	{
		number_of_bonds_central_[atom_i] = createBondsList(atom_i, bonds_list_central_[atom_i]);
		n_i_total = computeElectronDensity(number_of_bonds_central_[atom_i], bonds_list_central_[atom_i]);
		Uprime_[atom_i] = U_spline_->evaluate_deriv(n_i_total);
	}

    int max_number_of_central_forces = 0;

    for (atom_i = 0; atom_i < number_of_atoms_; atom_i++)
    {
        tmp_number_of_central_forces = 0;

        number_of_bonds_i = number_of_bonds_central_[atom_i];
        bonds_list_i = bonds_list_central_[atom_i];

// sprytna lista sasiadow - aktualizacja
        for (j = 0; j < n_num_[atom_i]; j++)
        {
            atom_j = n_list_[atom_i][j];
            memcpy(&smart_neighbours_list_[atom_j][0], &atom_i, sizeof(int));
            memcpy(&smart_neighbours_list_[atom_j][1], &j, sizeof(int));
        }

// start: sily centralne typu NN
        for (i = 0; i < number_of_bonds_i; i++)
        {
            f_r_ij = bonds_list_i[i].f_r_ij;

            if ( f_r_ij != 0.0 )
            {
                atom_j = bonds_list_i[i].atom_j;
                r_ij = bonds_list_i[i].r_ij;
                inv_of_r_ij = bonds_list_i[i].inv_of_r_ij;
                r_ij_dir[0] = bonds_list_i[i].r_ij_dir[0];
                r_ij_dir[1] = bonds_list_i[i].r_ij_dir[1];
                r_ij_dir[2] = bonds_list_i[i].r_ij_dir[2];
                fprime_r_ij = bonds_list_i[i].fprime_r_ij;
				
// obliczanie Theta1_jik, Theta2_jik, Theta3_jik
                Theta1_jik = 0.0;
                Theta2_jik = 0.0;
                Theta3_jik = 0.0;
                for (j = 0; j < number_of_bonds_i; j++)
                    if ( j != i )
                    {
                        f_r_ik = bonds_list_i[j].f_r_ij;

                        if ( f_r_ik == 0.0 )
                            continue;

                        inv_of_r_ik = bonds_list_i[j].inv_of_r_ij;
                        r_ik_dir[0] = bonds_list_i[j].r_ij_dir[0];
                        r_ik_dir[1] = bonds_list_i[j].r_ij_dir[1];
                        r_ik_dir[2] = bonds_list_i[j].r_ij_dir[2];

                        computeCosThetaJIK(r_ij_dir, r_ik_dir, cos_theta_jik);
                        g_cos_theta_jik = g_spline_->evaluate(cos_theta_jik, gprime_cos_theta_jik);

                        Theta1_jik += ( f_r_ik * g_cos_theta_jik );
                        Theta2_jik += ( inv_of_r_ik * f_r_ik * gprime_cos_theta_jik );
                        Theta3_jik += ( f_r_ik * gprime_cos_theta_jik * cos_theta_jik );
                    }

// obliczanie Theta1_ijk, Theta2_ijk, Theta3_ijk
                Theta1_ijk = 0.0;
                Theta2_ijk = 0.0;
                Theta3_ijk = 0.0;

                number_of_bonds_j = number_of_bonds_central_[atom_j];
                bonds_list_j = bonds_list_central_[atom_j];
                for (j = 0; j < number_of_bonds_j; j++)
                {
                    atom_k = bonds_list_j[j].atom_j;
                    if ( atom_k == atom_i )
                        continue;

                    f_r_jk = bonds_list_j[j].f_r_ij;
                    if ( f_r_jk == 0.0 )
                        continue;

                    inv_of_r_jk = bonds_list_j[j].inv_of_r_ij;
                    r_jk_dir[0] = bonds_list_j[j].r_ij_dir[0];
                    r_jk_dir[1] = bonds_list_j[j].r_ij_dir[1];
                    r_jk_dir[2] = bonds_list_j[j].r_ij_dir[2];

                    computeCosThetaJIK(r_jk_dir, r_ij_dir, cos_theta_kji);
                    cos_theta_kji *= -1.0;
                    g_cos_theta_kji = g_spline_->evaluate(cos_theta_kji, gprime_cos_theta_kji);

                    Theta1_ijk += ( f_r_jk * g_cos_theta_kji );
                    Theta2_ijk += ( inv_of_r_jk * f_r_jk * gprime_cos_theta_kji );
                    Theta3_ijk += ( f_r_jk * gprime_cos_theta_kji * cos_theta_kji );
                }

                prefactor  = Uprime_[atom_i] * ( fprime_r_ij * Theta1_jik + f_r_ij * Theta2_jik - f_r_ij * inv_of_r_ij * Theta3_jik );
                prefactor += Uprime_[atom_j] * ( fprime_r_ij * Theta1_ijk + f_r_ij * Theta2_ijk - f_r_ij * inv_of_r_ij * Theta3_ijk );
            }
            else
            {
                prefactor = 0.0;
                atom_j = bonds_list_i[i].atom_j;
                r_ij = bonds_list_i[i].r_ij;
                r_ij_dir[0] = bonds_list_i[i].r_ij_dir[0];
                r_ij_dir[1] = bonds_list_i[i].r_ij_dir[1];
                r_ij_dir[2] = bonds_list_i[i].r_ij_dir[2];
                number_of_bonds_j = number_of_bonds_central_[atom_j];
                bonds_list_j = bonds_list_central_[atom_j];
            }

            phi_prime_r_ij = phi_spline_->evaluate_deriv(r_ij);
            prefactor += phi_prime_r_ij;

            rho_prime_r_ij = rho_spline_->evaluate_deriv(r_ij);
            prefactor += ( rho_prime_r_ij * ( Uprime_[atom_i] + Uprime_[atom_j] ) );

            tmp_central_force[0] = prefactor * r_ij_dir[0];
            tmp_central_force[1] = prefactor * r_ij_dir[1];
            tmp_central_force[2] = prefactor * r_ij_dir[2];

            compute_Theta4_kji(atom_i, number_of_bonds_j,
                               bonds_list_i, bonds_list_j,
                               Theta4_kji);

            tmp_central_force[0] -= Theta4_kji * r_ij_dir[0] * r_ij;
            tmp_central_force[1] -= Theta4_kji * r_ij_dir[1] * r_ij;
            tmp_central_force[2] -= Theta4_kji * r_ij_dir[2] * r_ij;

            if ( tmp_number_of_central_forces == max_number_of_central_forces_ )
                raiseError(ERR_CFD_SMEAM, 2, "compute_central_forces", "too many central forces");

            this_central_force = &central_forces_[atom_i][tmp_number_of_central_forces];
            this_central_force->first_second = 0;
            this_central_force->atom_j = atom_j;
            this_central_force->force[0] = tmp_central_force[0];
            this_central_force->force[1] = tmp_central_force[1];
            this_central_force->force[2] = tmp_central_force[2];
            this_central_force->r_ij = r_ij;
            this_central_force->r_ij_dir[0] = r_ij_dir[0];
            this_central_force->r_ij_dir[1] = r_ij_dir[1];
            this_central_force->r_ij_dir[2] = r_ij_dir[2];
            tmp_number_of_central_forces++;
        }
// stop: sily centralne typu NN

// uwaga: poprzez czlon Theta4_kji z atomem i-tym oddzialywuja rowniez pozostale atomy ukladu,
//        w powyzszej petli obliczylismy juz sily dla atomow bedacych najblizszymi sasiadami atomu i-tego,
//        wymagane jest jeszcze obliczenie pozostalych sil ,,trojcialowych''
//        pomiedzy atomem i-tym a atomami, ktore nie sa jego bezposrednimi sasiadami,
//        atomy takie charakteryzuja sie tym, iz posiadaja razem z atomem i-tym co najmniej jednego wspolnego sasiada

// start: sily centralne typu non-NN
        for (i = 0; i < s_num_[atom_i]; i++)
        {
            atom_j = s_list_[atom_i][i];
            number_of_bonds_j = number_of_bonds_central_[atom_j];
            bonds_list_j = bonds_list_central_[atom_j];
            nonzero_force = compute_Theta4_kji(atom_i, number_of_bonds_j,
                                               bonds_list_i, bonds_list_j,
                                               Theta4_kji);

            if ( nonzero_force == 1 )
            {
                bond_r_ij = &s_bonds_[atom_i][i];
                r_ij = bond_r_ij->r;
                inv_of_r_ij = 1.0 / r_ij;
                r_ij_dir[0] = bond_r_ij->vec[0] * inv_of_r_ij;
                r_ij_dir[1] = bond_r_ij->vec[1] * inv_of_r_ij;
                r_ij_dir[2] = bond_r_ij->vec[2] * inv_of_r_ij;

                r_ij_vec[0] = bond_r_ij->vec[0];
                r_ij_vec[1] = bond_r_ij->vec[1];
                r_ij_vec[2] = bond_r_ij->vec[2];
                tmp_central_force[0] = - Theta4_kji * r_ij_vec[0];
                tmp_central_force[1] = - Theta4_kji * r_ij_vec[1];
                tmp_central_force[2] = - Theta4_kji * r_ij_vec[2];

                if ( tmp_number_of_central_forces == max_number_of_central_forces_ )
                    raiseError(ERR_CFD_SMEAM, 3, "compute_central_forces", "too many central forces");

                this_central_force = &central_forces_[atom_i][tmp_number_of_central_forces];
                this_central_force->first_second = 1;
                this_central_force->atom_j = atom_j;
                this_central_force->force[0] = tmp_central_force[0];
                this_central_force->force[1] = tmp_central_force[1];
                this_central_force->force[2] = tmp_central_force[2];
                this_central_force->r_ij = r_ij;
                this_central_force->r_ij_dir[0] = r_ij_dir[0];
                this_central_force->r_ij_dir[1] = r_ij_dir[1];
                this_central_force->r_ij_dir[2] = r_ij_dir[2];
                tmp_number_of_central_forces++;
            }
        }
// stop: sily centralne typu non-NN

        number_of_central_forces_[atom_i] = tmp_number_of_central_forces;
        if ( tmp_number_of_central_forces > max_number_of_central_forces )
            max_number_of_central_forces = tmp_number_of_central_forces;
    }

    #ifdef CFD_SMEAM_DEBUG
    std::cout << std::endl;
    std::cout << "*** CFD_sMEAM::compute_central() done! ***" << std::endl;
    std::cout << std::endl;
    #endif

    #ifdef CFD_SMEAM_TIMINGS
    timer_->routineStopped(3);
    #endif
}


void CUDA_CFD_sMEAM::write_central_foces(std::string file_name)
{
    #ifdef CFD_SMEAM_TIMINGS
    timer_->routineStarted(4);
    #endif

    CudaCentralForcesFile *cforces_file;

	cforces_file = new CudaCentralForcesFile();
    cforces_file->openOutFile(file_name);
    cforces_file->writeToOutFile(number_of_atoms_, max_number_of_central_forces_,
                                 number_of_central_forces_, central_forces_);
    cforces_file->closeOutFile();
    delete cforces_file;

    #ifdef CFD_SMEAM_TIMINGS
    timer_->routineStopped(4);
    #endif
}
